#include "hip/hip_runtime.h"
//                         _ooOoo_
//                        o8888888o
//                        88" . "88
//                        (| -_- |)
//                        O\  =  /O
//                     ____/'---'\____
//                   .'  \\|     |//  '.
//                  /  \\|||  :  |||//  \
//                 /  _||||| -:- |||||-  \
//                 |   | \\\  _  /// |   |
//                 | \_|  ''\---/''  |   |
//                 \  .-\__  '_'  ___/-. /
//               ___`. .'  /--.--\  '. . __
//            ."" '<  `.___\_<|>_/___.'  >'"".
//           | | :  `- \`.;`\ _ /`;.`/ - ` : | |
//           \  \ `-.   \_ __\ /__ _/   .-`  / /
// ===========`-.____`-.___\_____/___.-`_____.-`================
//                          `=---='
// ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
// =============================================================

//152行不能整除，整除出bug
//kernel_size不能超过block_size_x和block_size_y，不然会出现越界的问题
#include <opencv2/opencv.hpp>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hip/device_functions.h"
#include <malloc.h>
/*//#include <helper_gl.h>
#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>
#include <helper_functions.h>
#include <GL/freeglut.h>
#include <hip/hip_vector_types.h>
#include <driver_functions.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <GL/glx.h>*/

#define uint unsigned int
#define uchar unsigned char
using namespace std;
using namespace cv;
#define block_size_x 8
#define block_size_y 64

//new define sum area
//kernel_size>=2*box_kernel+1!!
#define box_kernel 3
#define MIDDLE_kernel 0.09//调节加权中值滤波取的第几个值,配合sigma_c
#define MIDDLE_kernel_h 0.8
#define error_range 25
#define Middle_Radius 3
#define filterweight_kernel 3//size = weightedmedian_kernel*2+1
#define filterweight_kernel_h 2//size = weightedmedian_kernel*2+1
#define kernel_size box_kernel*2+1
#define min(x,y)  ( x<y?x:y )
#define max(x,y)  ( x<y?y:x )

#define rows 992   //948*1500
#define cols 1420
#define disp_max 160
#define scale_pic 1
#define sigma_c 8//weightedmedianfilter parameter
#define sigma_p 16
#define r1 5
#define step 1

uchar (*temp)[cols];

//int shift=(kernel_size+1)/2;

int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__global__ void Match_error(uchar (*a)[cols],uchar (*b)[cols],bool (*c)[cols])
{
	const uint idx=(blockIdx.x*blockDim.x)+threadIdx.x;
	const uint idy=(blockIdx.y*blockDim.y)+threadIdx.y;
	if((idy>disp_max+(kernel_size-1)/2+box_kernel)&&(idy<cols-disp_max-(kernel_size-1)/2-box_kernel)&&(idx>(kernel_size-1)/2+box_kernel)&&(idx<rows-(kernel_size-1)/2-box_kernel))
	{
		int a0=a[idx][idy];
		uint yy=idy-a0;
		int b0=b[idx][yy];
		c[idx][idy]=((a0-b0)<error_range)?0:1;
	}
//	a[idx][idy]=(uchar)a[idx][idy];
}

__global__ void near_match(	uchar (*a)[cols],uchar (*a1)[cols],bool (*b)[cols],uchar (*r)[cols],
							uchar (*l)[cols],uchar (*u)[cols],uchar (*d)[cols],
							uchar (*lr)[cols],uchar (*ud)[cols])
//a是视察图，b是匹配错误分析图
{
	const uint idx=(blockIdx.x*blockDim.x)+threadIdx.x;
	const uint idy=(blockIdx.y*blockDim.y)+threadIdx.y;
	r[idx][idy]=0;
	l[idx][idy]=0;
	u[idx][idy]=0;
	d[idx][idy]=0;
	lr[idx][idy]=0;
	ud[idx][idy]=0;
	a1[idx][idy]=a[idx][idy];
	int win_move=(r1-1)/2+1;
	uint k;
	for (k=0; k<win_move; k++)
	{
		if(b[idx][idy+k]==0&(idy+k)<cols)
			{
			r[idx][idy]=a[idx][idy+k];
			break;
			}
	}
	for (k=0; k<win_move; k++)
	{
		int T=idy-k;
		if(b[idx][T]==0&(T>0))
		{
			l[idx][idy]=a[idx][T];
			break;
		}
	}
	for (k=0; k<win_move; k++)
	{
		int T=idx-k;
		if(b[T][idy]==0 &(T)>0)
		{
			u[idx][idy]=a[T][idy];
			break;
		}
	}

	for (k=0; k<win_move; k++)
	{
		if(b[idx+k][idy]==0 &(idx+k)<cols)
		{
			d[idx][idy]=a[idx+k][idy];
			break;
		}
	}
	if(b[idx][idy]==1)
	{
		if ((r[idx][idy]==0)&&(l[idx][idy]>0))
		{
			lr[idx][idy]=l[idx][idy];
		}
		if ((r[idx][idy]>0)&&(l[idx][idy]==0))
		{
			lr[idx][idy]=r[idx][idy];
		}
		if ((r[idx][idy]>0)&&(l[idx][idy]>0))
		{
			lr[idx][idy]=min(r[idx][idy],l[idx][idy]);
		}

		if ((u[idx][idy]==0)&&(d[idx][idy]>0))
		{
			ud[idx][idy]=d[idx][idy];
		}
		if ((u[idx][idy]>0)&&(d[idx][idy]==0))
		{
			ud[idx][idy]=u[idx][idy];
		}
		if ((u[idx][idy]>0)&&(d[idx][idy]>0))
		{
			ud[idx][idy]=min(u[idx][idy],d[idx][idy]);
		}
		if ((ud[idx][idy]>0)&&(lr[idx][idy]==0))
		{
			a1[idx][idy]=ud[idx][idy];
			b[idx][idy]=0;
		}
		if ((ud[idx][idy]==0)&&(lr[idx][idy]>0))
		{
			a1[idx][idy]=lr[idx][idy];
			b[idx][idy]=0;
		}
		if ((ud[idx][idy]>0)&&(lr[idx][idy]>0))
		{
			a1[idx][idy]=min(lr[idx][idy],ud[idx][idy]);
			b[idx][idy]=0;
		}
	}
}

/*
 __global__ void stereo_kernel(uint (*a)[cols],uint (*b)[cols],uchar (*disp)[cols])
{
	const uint x=(blockIdx.x*blockDim.x)+threadIdx.x;
	const uint y=(blockIdx.y*blockDim.y)+threadIdx.y;
	__shared__ uint sh_left[block_size_x][block_size_y];
	__shared__ uint sh_right[block_size_x][block_size_y+disp_max];
	int y1=blockIdx.y*blockDim.y+threadIdx.y-disp_max;
	int y2=blockIdx.y*blockDim.y+threadIdx.y+64-disp_max;
	int y3=blockIdx.y*blockDim.y+threadIdx.y+128-disp_max;
	if(y<disp_max)
	{
		disp[x][y]=0;
	}
	else
	{
		sh_left[threadIdx.x][threadIdx.y]=a[x][y];
		if(threadIdx.y<32)
		{
			sh_right[threadIdx.x][threadIdx.y]         =b[x][y1];
			sh_right[threadIdx.x][threadIdx.y+64]      =b[x][y2];
			sh_right[threadIdx.x][threadIdx.y+128]     =b[x][y3];
			sh_right[threadIdx.x][threadIdx.y+disp_max]=b[x][y];
		}
		else
		{
			sh_right[threadIdx.x][threadIdx.y]         =b[x][y1];
			sh_right[threadIdx.x][threadIdx.y+64]      =b[x][y2];
			sh_right[threadIdx.x][threadIdx.y+disp_max]=b[x][y];
		}
		__syncthreads();
		disp[x][y]=0;
		uint cost=abs((float)(sh_left[threadIdx.x][threadIdx.y]-sh_right[threadIdx.x][threadIdx.y+disp_max])); //(float)(b[x][y])
		uint cost_now;
		for(int d=1;d<disp_max+1;d+=step)
		{
			cost_now=abs((float)(sh_left[threadIdx.x][threadIdx.y]-sh_right[threadIdx.x][threadIdx.y+(disp_max-d)]));  //(float)(b[x][y-d])
			if(cost>cost_now)
			{
				disp[x][y]=d;
				cost=cost_now;
			}
		}
	}
}
 */

__global__ void stereo_kernel(uint (*a)[cols],uint (*b)[cols],uchar (*disp)[cols])
{
	const uint x=(blockIdx.x*blockDim.x)+threadIdx.x;
	const uint y=(blockIdx.y*blockDim.y)+threadIdx.y;
	//test0=10;
	//因为用的是3X3的区域内计算，所以上下左右方向各扩充一格

	__shared__ uint sh_left[block_size_x+kernel_size-1+box_kernel*2+1][block_size_y+kernel_size-1+box_kernel*2+1];//共享内存为什么要大点，共享内存只能在方格内部起作用
	__shared__ uint sh_right[block_size_x+kernel_size-1+box_kernel*2+1][block_size_y+disp_max+kernel_size-1+box_kernel*2+1];

	//这些int都相当于寄存器
	//int y1=blockIdx.y*blockDim.y+threadIdx.y-disp_max;//这个定义中64,128还有就是disp_max作用？0-64
	//int y2=blockIdx.y*blockDim.y+threadIdx.y+64-disp_max;//block_size_y的大小是64,视差大小是160,所以要3个64，64-128
	//int y3=blockIdx.y*blockDim.y+threadIdx.y+128-disp_max;//128-192

	if((y>disp_max+(kernel_size-1)/2+box_kernel)&&(y<cols-(kernel_size-1)/2-box_kernel)&&(x>(kernel_size-1)/2+box_kernel)&&(x<rows-(kernel_size-1)/2-box_kernel))//y>disp_max，表示了左边disp_max个点不要了
	{
		//configure left shared memory
		sh_left[threadIdx.x][threadIdx.y]=a[x-(kernel_size-1)/2][y-(kernel_size-1)/2];//因为共享内存上下左右各大了一行，而坐标原点在左上角，所以需要全部-1
		if(threadIdx.x<(kernel_size-1+box_kernel))//内存重复覆盖的问题？这里这个threadIdx.x是不是一定小于2,因为block_size_x是2，所以判断语句是不是不需要了？
		{
			sh_left[threadIdx.x+block_size_x][threadIdx.y]=a[x+block_size_x-(kernel_size-1)/2][y-(kernel_size-1)/2];//相当于存了4行的数据，上下各拓宽了一行
		}
		if(threadIdx.y<(kernel_size-1+box_kernel))//这个是为了扩充右边界，因为上下左右各拓宽了一行
		{
			sh_left[threadIdx.x][threadIdx.y+block_size_y]=a[x-(kernel_size-1)/2][y+block_size_y-(kernel_size-1)/2];
		}
		if((threadIdx.x<(kernel_size-1+box_kernel))&&(threadIdx.y<(kernel_size-1+box_kernel)))//这个是为了更新边框右下角的数据
		{
			sh_left[threadIdx.x+block_size_x][threadIdx.y+block_size_y]=a[x+block_size_x-(kernel_size-1)/2][y+block_size_y-(kernel_size-1)/2];
		}

		//configure right shared memory
		//y和x不同，y需要考虑视差160的因素，所以需要3个blocks才能计算一个视差出来，如果恰巧在边界上应该计算4个blocks？
		int cacl_number=disp_max-block_size_y*(disp_max/block_size_y);//这个地方不能整除，整除出bug
		int cacl_Num=(disp_max+block_size_y-1)/block_size_y;
		if(threadIdx.y<(cacl_number))//在0-32的范围内，64×3=32+160,正好只需要3个blocks就能够完成计算
		{
			for(int i=0;i<cacl_Num;i++)
			{
				sh_right[threadIdx.x][threadIdx.y+block_size_y*i] = b[x-(kernel_size-1)/2][y-disp_max-(kernel_size-1)/2+block_size_y*i];
			}
			sh_right[threadIdx.x][threadIdx.y+disp_max] = b[x-(kernel_size-1)/2][y-(kernel_size-1)/2];
			//sh_right[threadIdx.x][threadIdx.y]         =b[x-(kernel_size-1)/2][y1-(kernel_size-1)/2];//正常的第一个blocks0-64，这里是0-31的视差
			//sh_right[threadIdx.x][threadIdx.y+64]      =b[x-1][y2-1];//第二个blocks64-128，这里是64-95的视差
			//sh_right[threadIdx.x][threadIdx.y+128]     =b[x-1][y3-1];//第三个blocks128-192，这里是128-159的视差
			//sh_right[threadIdx.x][threadIdx.y+disp_max]=b[x-1][y-1];//这里是160-191的视差范围，这些点虽然不算，但是计算时需要用到
			if(threadIdx.x<(kernel_size-1))//配置下边界的数据
			{
				for(int i=0;i<cacl_Num;i++)
				{
					sh_right[threadIdx.x+block_size_x][threadIdx.y+block_size_y*i] = b[x+block_size_x-(kernel_size-1)/2][y-disp_max-(kernel_size-1)/2+block_size_y*i];
				}
				sh_right[threadIdx.x+block_size_x][threadIdx.y+disp_max]=b[x+block_size_x-(kernel_size-1)/2][y-(kernel_size-1)/2];
				//sh_right[threadIdx.x+block_size_x][threadIdx.y]         =b[x+block_size_x-(kernel_size-1)/2][y1-(kernel_size-1)/2];
				//sh_right[threadIdx.x+2][threadIdx.y+64]      =b[x+1][y2-1];
				//sh_right[threadIdx.x+2][threadIdx.y+128]     =b[x+1][y3-1];
				//sh_right[threadIdx.x+2][threadIdx.y+disp_max]=b[x+1][y-1];
			}
			if(threadIdx.y<(kernel_size-1))//配置右边界数据，这里的224是64+160,这个if的位置有所改动
			{
				sh_right[threadIdx.x][threadIdx.y+block_size_y+disp_max]=b[x-(kernel_size-1)/2][y+block_size_y-(kernel_size-1)/2];//这里实在是太牛逼了，
			}//																			//blockIdx.y*blockDim.y相当于前一个block的最后一个数据
			if((threadIdx.x<(kernel_size-1))&&(threadIdx.y<(kernel_size-1)))//这里配置右下角
			{
				sh_right[threadIdx.x+block_size_x][threadIdx.y+block_size_y+disp_max]=b[x+block_size_x-(kernel_size-1)/2][y+block_size_y-(kernel_size-1)/2];//这个地方还是有个问题，将一个b值付给了2个thread值
			}
		}
		else//当threadIdx.y>32时，需要4个blocks才能完成计算，因为64*3<threadIdx.y+160
		{
			for(int i=0;i<(cacl_Num-1);i++)
			{
				sh_right[threadIdx.x][threadIdx.y+block_size_y*i] = b[x-(kernel_size-1)/2][y-disp_max-(kernel_size-1)/2+block_size_y*i];
			}
			sh_right[threadIdx.x][threadIdx.y+disp_max] = b[x-(kernel_size-1)/2][y-(kernel_size-1)/2];
			//sh_right[threadIdx.x][threadIdx.y]         =b[x-1][y1-1];//这里64>threadIdx.y>32的视差
			//sh_right[threadIdx.x][threadIdx.y+64]      =b[x-1][y2-1];//这里是96-127的视差
			//sh_right[threadIdx.x][threadIdx.y+disp_max]=b[x-1][y-1];//这里是192-223的视差
			if(threadIdx.x<(kernel_size-1))//配置对应的下边界
			{
				for(int i=0;i<(cacl_Num-1);i++)
				{
					sh_right[threadIdx.x+block_size_x][threadIdx.y+block_size_y*i] = b[x+block_size_x-(kernel_size-1)/2][y-disp_max-(kernel_size-1)/2+block_size_y*i];
				}
				sh_right[threadIdx.x+block_size_x][threadIdx.y+disp_max]=b[x+block_size_x-(kernel_size-1)/2][y-(kernel_size-1)/2];
				//sh_right[threadIdx.x+2][threadIdx.y]         =b[x+1][y1-1];
				//sh_right[threadIdx.x+2][threadIdx.y+64]      =b[x+1][y2-1];
				//sh_right[threadIdx.x+2][threadIdx.y+disp_max]=b[x+1][y-1];
			}
			if(threadIdx.y<(kernel_size-1))//配置对应的右边界
			{
				sh_right[threadIdx.x][threadIdx.y+block_size_y+disp_max]=b[x-(kernel_size-1)/2][y+block_size_y-(kernel_size-1)/2];
				//sh_right[threadIdx.x][threadIdx.y+224]=b[x-1][blockIdx.y*blockDim.y+63];
			}
			if((threadIdx.x<(kernel_size-1))&&(threadIdx.y<(kernel_size-1)))//配置右下边界
			{
				sh_right[threadIdx.x+block_size_x][threadIdx.y+block_size_y+disp_max]=b[x+block_size_x-(kernel_size-1)/2][y+block_size_y-(kernel_size-1)/2];
				//sh_right[threadIdx.x+2][threadIdx.y+224]=b[x+1][blockIdx.y*blockDim.y+63];
			}
		}
		// temp1存0-8位，temp2存9-15位，temp3存16-24位
		int model1=255;
		int model2=255*2^8;
		int temp1_l=0;
		int temp1_r=0;
		int temp2_l=0;
		int temp2_r=0;
		int temp3_l=0;
		int temp3_r=0;
		__syncthreads();
		disp[x][y]=0;
		int cost=0;
		for(int i=(-box_kernel);i<(box_kernel+1);i++)
		{
//		int i =0;
			for (int j=(-box_kernel);j<(box_kernel+1);j++)
			{
				temp1_l=((int)sh_left[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+(kernel_size-1)/2+j])&model1;
				temp1_r=((int)sh_right[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+disp_max+(kernel_size-1)/2+j])&model1;
				temp2_l=((int)sh_left[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+(kernel_size-1)/2+j])&model2;
				temp2_l=temp2_l>>8;
				temp2_r=((int)sh_right[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+disp_max+(kernel_size-1)/2+j])&model2;
				temp2_r=temp2_r>>8;
				temp3_l=((int)sh_left[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+(kernel_size-1)/2+j])>>16;
				temp3_r=((int)sh_right[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+disp_max+(kernel_size-1)/2+j])>>16;
				cost+=abs(temp1_l-temp1_r)+abs(temp2_l-temp2_r)+abs(temp3_l-temp3_r);
//				cost+=abs((int)sh_left[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+(kernel_size-1)/2+j]-(int)sh_right[threadIdx.x+(kernel_size-1)/2+i][threadIdx.y+disp_max+(kernel_size-1)/2+j]);
			}
		}
		//
//		cost=		(//abs((float)(sh_left[threadIdx.x][threadIdx.y]-sh_right[threadIdx.x][threadIdx.y+disp_max]))
//						+abs((float)sh_left[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(kernel_size-1)/2]-(float)sh_right[threadIdx.x+(kernel_size-1)/2][threadIdx.y+disp_max+(kernel_size-1)/2])
						//+abs((float)(sh_left[threadIdx.x][threadIdx.y+2]-sh_right[threadIdx.x][threadIdx.y+disp_max+2]))
					//	+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2-1][threadIdx.y+(kernel_size-1)/2]-sh_right[threadIdx.x+(kernel_size-1)/2-1][threadIdx.y+disp_max+(kernel_size-1)/2]))
					//	+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2+1][threadIdx.y+(kernel_size-1)/2]-sh_right[threadIdx.x+(kernel_size-1)/2+1][threadIdx.y+disp_max+(kernel_size-1)/2]))
					//	+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(kernel_size-1)/2-1]-sh_right[threadIdx.x+(kernel_size-1)/2][threadIdx.y+disp_max+(kernel_size-1)/2-1]))
						//+abs((float)(sh_left[threadIdx.x+2][threadIdx.y]-sh_right[threadIdx.x+2][threadIdx.y+disp_max]))
					//	+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(kernel_size-1)/2+1]-sh_right[threadIdx.x+(kernel_size-1)/2][threadIdx.y+disp_max+(kernel_size-1)/2+1]))
//						/*+abs((float)(sh_left[threadIdx.x+2][threadIdx.y+2]-sh_right[threadIdx.x+2][threadIdx.y+disp_max+2]))*/)/9; //(float)(b[x][y])
		int cost_now;
		for(int d=1;d<disp_max+1;d+=step)
		{
			cost_now=0;

			for(int k=(-box_kernel);k<(box_kernel+1);k++)
			{
				for (int l=(-box_kernel);l<(box_kernel+1);l++)
				{
					temp1_l=((int)sh_left[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(kernel_size-1)/2+l])&model1;
					temp1_r=((int)sh_right[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(disp_max-d)+(kernel_size-1)/2+l])&model1;
					temp2_l=((int)sh_left[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(kernel_size-1)/2+l])&model2;
					temp2_l=temp2_l>>8;
					temp2_r=((int)sh_right[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(disp_max-d)+(kernel_size-1)/2+l])&model2;
					temp2_r=temp2_r>>8;
					temp3_l=((int)sh_left[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(kernel_size-1)/2+l])>>16;
					temp3_r=((int)sh_right[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(disp_max-d)+(kernel_size-1)/2+l])>>16;
					cost_now+=abs(temp1_l-temp1_r)+abs(temp2_l-temp2_r)+abs(temp3_l-temp3_r);
//					cost_now+=abs((int)sh_left[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(kernel_size-1)/2+l]-(int)sh_right[threadIdx.x+(kernel_size-1)/2+k][threadIdx.y+(disp_max-d)+(kernel_size-1)/2+l]);
				}
			}
//			for (int l=(-box_kernel);l<(box_kernel+1);l++)
//			{
//				cost_now+=abs(((int)sh_left[threadIdx.x+(kernel_size-1)/2-4][threadIdx.y+(kernel_size-1)/2+l]-(int)sh_right[threadIdx.x+(kernel_size-1)/2-4][threadIdx.y+(disp_max-d)+(kernel_size-1)/2+l]);
//			}
			//
//			cost_now=		(//abs((float)(sh_left[threadIdx.x][threadIdx.y]-sh_right[threadIdx.x][threadIdx.y+(disp_max-d)]))
//							+abs((float)sh_left[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(kernel_size-1)/2]-(float)sh_right[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(disp_max-d)+(kernel_size-1)/2])
							//+abs((float)(sh_left[threadIdx.x][threadIdx.y+2]-sh_right[threadIdx.x][threadIdx.y+(disp_max-d)+2]))
							//+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2-1][threadIdx.y+(kernel_size-1)/2]-sh_right[threadIdx.x+(kernel_size-1)/2-1][threadIdx.y+(disp_max-d)+(kernel_size-1)/2]))
						//	+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2+1][threadIdx.y+(kernel_size-1)/2]-sh_right[threadIdx.x+(kernel_size-1)/2+1][threadIdx.y+(disp_max-d)+(kernel_size-1)/2]))
							//+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(kernel_size-1)/2-1]-sh_right[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(disp_max-d)+(kernel_size-1)/2-1]))
							//+abs((float)(sh_left[threadIdx.x+2][threadIdx.y]-sh_right[threadIdx.x+2][threadIdx.y+(disp_max-d)]))
							//+abs((float)(sh_left[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(kernel_size-1)/2+1]-sh_right[threadIdx.x+(kernel_size-1)/2][threadIdx.y+(disp_max-d)+(kernel_size-1)/2+1]))
//							/*+abs((float)(sh_left[threadIdx.x+2][threadIdx.y+2]-sh_right[threadIdx.x+2][threadIdx.y+(disp_max-d)+2]))*/)/9;  //(float)(b[x][y-d])
			if(cost>cost_now)
			{
				disp[x][y]=d;
				cost=cost_now;
			}
		}
	}
	else
	{
		disp[x][y]=0;
	}
}

__global__ void box_x(uchar (*input)[cols],uchar (*output)[cols],int win_radius)
{
	const uint idx= (blockIdx.x*blockDim.x) + threadIdx.x;
	const uint idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	uint scale=(win_radius<<1)+1;
	if ((idx >= win_radius) && (idx < rows - 1 - win_radius) && (idy >= win_radius) && (idy < cols - 1 - win_radius))
	{
		uint sum=0;
		for (int x = idx-win_radius; x <idx+win_radius+1 ; x++)
		{
			sum += input[x][idy];
		}
		output[idx][idy]=sum/scale;
	}
	else
		output[idx][idy]=input[idx][idy];
}

__global__ void box_y(uchar (*input)[cols],uchar (*output)[cols],int win_radius)
{
	const uint idx= (blockIdx.x*blockDim.x) + threadIdx.x;
	const uint idy =(blockIdx.y*blockDim.y) + threadIdx.y;
	uint scale=(win_radius<<1)+1;
	if ((idx >= win_radius) && (idx < rows - 1 - win_radius) && (idy >= win_radius) && (idy < cols - 1 - win_radius))
	{
		uint sum=0;
		for (int y = idy-win_radius; y <idy+win_radius+1 ; y++)
		{
			sum += input[idx][y];
		}
		output[idx][idy]=sum/scale;
	}
	else
		output[idx][idy]=input[idx][idy];
}

__global__ void flip_kernel(uint (*input)[cols],uint (*output)[cols])
{
	const uint idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const uint idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	if(idy<cols)
	output[idx][idy]=input[idx][cols-1-idy];
}
__global__ void flip_kernel_char(uchar (*input)[cols],uchar (*output)[cols])
{
	const uint idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const uint idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	if(idy<cols)
	output[idx][idy]=input[idx][cols-1-idy];
}

void box_filter(uchar (*input)[cols],uchar (*output)[cols],int win_radius,dim3 grid_size,dim3 block_size)
{
	box_x<<<grid_size,block_size>>>(input,temp,win_radius);
	box_y<<<grid_size,block_size>>>(temp,output,win_radius);
}

__global__ void middle_filter(uchar (*input)[cols],uchar (*output)[cols],int middle_radius)
{
	const uint idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const uint idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	int temp[Middle_Radius*Middle_Radius];
	int i;
	int j;
	if((idx>(middle_radius-1)/2)&&idx<(rows-(kernel_size-1)/2-1)&&idy<(cols-1-(kernel_size-1)/2-disp_max)&&idy>(disp_max+(kernel_size-1)/2))
	{
		for(int i=-(middle_radius-1)/2;i<(middle_radius+1)/2;i++)
		{
			for(int j=-(middle_radius-1)/2;j<(middle_radius+1)/2;j++)
			{
				temp[(i+(middle_radius-1)/2)*middle_radius+(j+(middle_radius-1)/2)]=input[idx+i][idy+j];
			}
		}

		for (int j=0; j<(middle_radius+1); ++j)
		    {
		        int min=j;
		        for (int l=j+1; l<middle_radius*middle_radius; ++l)
		            if (temp[l] < temp[min])
		                min=l;
		        const float bbtemp=temp[j];
		        temp[j]=temp[min];
		        temp[min]=bbtemp;
		    }
		output[idx][idy]= temp[(middle_radius-1)/2];
	}
	else
	{
		output[idx][idy]=0;
	}
}
__global__ void pic_show(uchar (*input)[cols],uchar (*output)[cols])
{
	const uint idx = (blockIdx.x*blockDim.x) + threadIdx.x;
	const uint idy = (blockIdx.y*blockDim.y) + threadIdx.y;
	output[idx][idy]=input[idx][idy]*scale_pic;
}


__global__ void filter_weight(uint (*a)[cols],uchar (*b)[cols],uchar (*output)[cols],float (*temp_d)[2*filterweight_kernel+1])//filterweight_kernel
{
	//这里只要2*filterweight_kernel<block_size_x和block_size_y就行了
	const uint x=(blockIdx.x*blockDim.x)+threadIdx.x;
	const uint y=(blockIdx.y*blockDim.y)+threadIdx.y;
	output[x][y]=0;
	//test0=10;
	//因为用的是3X3的区域内计算，所以上下左右方向各扩充一格

	__shared__ uint sh_pic[block_size_x+filterweight_kernel*2][block_size_y+filterweight_kernel*2];//共享内存为什么要大点，共享内存只能在方格内部起作用
	__shared__ uint sh_disp[block_size_x+filterweight_kernel*2][block_size_y+filterweight_kernel*2];
	//这些int都相当于寄存器
	//int y1=blockIdx.y*blockDim.y+threadIdx.y-disp_max;//这个定义中64,128还有就是disp_max作用？0-64
	//int y2=blockIdx.y*blockDim.y+threadIdx.y+64-disp_max;//block_size_y的大小是64,视差大小是160,所以要3个64，64-128
	//int y3=blockIdx.y*blockDim.y+threadIdx.y+128-disp_max;//128-192
	//output[threadIdx.x][threadIdx.y]=0;
	if((y>disp_max+(kernel_size-1)/2+box_kernel)&&(y<cols-disp_max-(kernel_size-1)/2-box_kernel)&&(x>(kernel_size-1)/2+box_kernel+filterweight_kernel)&&(x<rows-(kernel_size-1)/2-box_kernel-filterweight_kernel))//y>disp_max，表示了左边disp_max个点不要了
	{
		//configure left shared memory
		sh_pic[threadIdx.x][threadIdx.y]=a[x-filterweight_kernel][y-filterweight_kernel];//因为共享内存上下左右各大了一行，而坐标原点在左上角，所以需要全部-1
		sh_disp[threadIdx.x][threadIdx.y]=b[x-filterweight_kernel][y-filterweight_kernel];
		if(threadIdx.x<(2*filterweight_kernel))//内存重复覆盖的问题？这里这个threadIdx.x是不是一定小于2,因为block_size_x是2，所以判断语句是不是不需要了？
		{
			sh_pic[threadIdx.x+block_size_x][threadIdx.y]=a[x+block_size_x-filterweight_kernel][y-filterweight_kernel];//相当于存了4行的数据，上下各拓宽了一行
			sh_disp[threadIdx.x+block_size_x][threadIdx.y]=b[x+block_size_x-filterweight_kernel][y-filterweight_kernel];
		}
		if(threadIdx.y<(2*filterweight_kernel))//这个是为了扩充右边界，因为上下左右各拓宽了一行
		{
			sh_pic[threadIdx.x][threadIdx.y+block_size_y]=a[x-filterweight_kernel][y+block_size_y-filterweight_kernel];
			sh_disp[threadIdx.x][threadIdx.y+block_size_y]=b[x-filterweight_kernel][y+block_size_y-filterweight_kernel];
		}
		if((threadIdx.x<2*filterweight_kernel)&&(threadIdx.y<2*filterweight_kernel))//这个是为了更新边框右下角的数据
		{
			sh_pic[threadIdx.x+block_size_x][threadIdx.y+block_size_y]=a[x+block_size_x-filterweight_kernel][y+block_size_y-filterweight_kernel];
			sh_disp[threadIdx.x+block_size_x][threadIdx.y+block_size_y]=b[x+block_size_x-filterweight_kernel][y+block_size_y-filterweight_kernel];
		}
		__syncthreads();

		float temp_pic_c[2*filterweight_kernel+1][2*filterweight_kernel+1];
		float temp_pic_a[2*filterweight_kernel+1][2*filterweight_kernel+1];//color+distance=all,temp_pic_d
		float temp_disp[2*filterweight_kernel+1][2*filterweight_kernel+1];

		int model1=255;
		int model2=255*2^8;
		int temp1_l=0;
		int temp1_r=0;
		int temp2_l=0;
		int temp2_r=0;
		int temp3_l=0;
		int temp3_r=0;


		temp1_r=(sh_pic[threadIdx.x+filterweight_kernel][threadIdx.y+filterweight_kernel])&model1;//这里1r存的是中心点的低8位值
		temp2_r=(sh_pic[threadIdx.x+filterweight_kernel][threadIdx.y+filterweight_kernel])&model2;//这里2r存的是中心点的中8位值
		temp2_r=temp2_r>>8;
		temp3_r=(sh_pic[threadIdx.x+filterweight_kernel][threadIdx.y+filterweight_kernel])>>16;//这里3r存的是中心点的高8位值
		for(int i=0;i<(2*filterweight_kernel+1);i++)
		{
			for(int j=0;j<(2*filterweight_kernel+1);j++)//关于颜色的地方需要改改！！RGB
			{
				temp1_l=(sh_pic[threadIdx.x+i][threadIdx.y+j])&model1;
				temp2_l=(sh_pic[threadIdx.x+i][threadIdx.y+j])&model2;
				temp2_l=temp2_l>>8;
				temp3_l=(sh_pic[threadIdx.x+i][threadIdx.y+j])>>16;
				temp_pic_c[i][j]=sqrt((float)((temp1_l-temp1_r)*(temp1_l-temp1_r)+(temp2_l-temp2_r)*(temp2_l-temp2_r)+(temp3_l-temp3_r)*(temp3_l-temp3_r)));
			}
		}
		for(int i=0;i<(2*filterweight_kernel+1);i++)
		{
			for(int j=0;j<(2*filterweight_kernel+1);j++)
			{
				temp_pic_a[i][j]=exp(-1*(temp_pic_c[i][j]/sigma_c)*(temp_pic_c[i][j]/sigma_c))*exp(-1*(temp_d[i][j]/sigma_p)*(temp_d[i][j]/sigma_p));//sigma_c和sigma_p需要定义
			}
		}
//		for(int i=0;i<(2*filterweight_kernel+1);i++)
//		{
//			for(int j=0;j<(2*filterweight_kernel+1);j++)
//			{
//				temp_disp[i][j]=temp_pic_a[i][j]*(float)sh_disp[threadIdx.x+i][threadIdx.y+j];//sigma_c和sigma_p需要定义
//			}
//		}
		//Middle filter
		float temp_disp_middle[(2*filterweight_kernel+1)*(2*filterweight_kernel+1)];
		float temp_pic_middle[(2*filterweight_kernel+1)*(2*filterweight_kernel+1)];
		for(int i=0;i<(2*filterweight_kernel+1);i++)
		{
			for(int j=0;j<(2*filterweight_kernel+1);j++)
			{
				temp_disp_middle[i*(2*filterweight_kernel+1)+j]=(float)sh_disp[threadIdx.x+i][threadIdx.y+j];
				temp_pic_middle[i*(2*filterweight_kernel+1)+j]=temp_pic_a[i][j];
			}
		}

		for (int j=0; j<(2*filterweight_kernel+1); ++j)
		    {
		        int min_middle=j;
		        for (int l=j+1; l<(2*filterweight_kernel+1)*(2*filterweight_kernel+1)-1; ++l)
		            if (temp_disp_middle[l] < temp_disp_middle[min_middle])
		            	min_middle=l;
		        //视差排序同时权值排序
		        const float bbtemp=temp_disp_middle[j];
		        temp_disp_middle[j]=temp_disp_middle[min_middle];
		        temp_disp_middle[min_middle]=bbtemp;

		        const float bbbtemp=temp_pic_middle[j];
		        temp_pic_middle[j]=temp_pic_middle[min_middle];
		        temp_pic_middle[min_middle]=bbbtemp;
		    }
		float SUM=0;
		for (int i=0;i<(2*filterweight_kernel+1)*(2*filterweight_kernel+1);i++)
		{
			SUM= SUM + temp_pic_middle[i];
		}
		SUM=SUM*MIDDLE_kernel;
		float Mid_SUM=0;
		for (int i=0;i<(2*filterweight_kernel+1);i++)
		{
			for (int j=0;j<(2*filterweight_kernel+1);j++)
			{
				Mid_SUM= Mid_SUM + temp_pic_middle[j+(2*filterweight_kernel+1)*i];
				if(Mid_SUM>=SUM)
				{
					output[x][y]= temp_disp_middle[j+(2*filterweight_kernel+1)*i];
					i=2*filterweight_kernel+1;
					break;
				}
			}
		}

//		for(int i=0;i<(2*filterweight_kernel+1); ++i)
//		{
//			for(int j=0;j<(2*filterweight_kernel+1);j++)
//			{
//				if(temp_disp[i][j]==temp_middle[filterweight_kernel])
//				{
//					output[x][y]= sh_disp[i+threadIdx.x][j+threadIdx.y];
//					i=2*filterweight_kernel+1;
//					break;
//				}
//			}
//		}
	}
}


__global__ void filter_weight_h(uint (*a)[cols],uchar (*b)[cols],uchar (*output)[cols],float (*temp_d)[2*filterweight_kernel_h+1])//filterweight_kernel_h
{
	//这里只要2*filterweight_kernel_h<block_size_x和block_size_y就行了
	const uint x=(blockIdx.x*blockDim.x)+threadIdx.x;
	const uint y=(blockIdx.y*blockDim.y)+threadIdx.y;
	output[x][y]=0;
	//test0=10;
	//因为用的是3X3的区域内计算，所以上下左右方向各扩充一格

	__shared__ uint sh_pic[block_size_x+filterweight_kernel_h*2][block_size_y+filterweight_kernel_h*2];//共享内存为什么要大点，共享内存只能在方格内部起作用
	__shared__ uint sh_disp[block_size_x+filterweight_kernel_h*2][block_size_y+filterweight_kernel_h*2];
	//这些int都相当于寄存器
	//int y1=blockIdx.y*blockDim.y+threadIdx.y-disp_max;//这个定义中64,128还有就是disp_max作用？0-64
	//int y2=blockIdx.y*blockDim.y+threadIdx.y+64-disp_max;//block_size_y的大小是64,视差大小是160,所以要3个64，64-128
	//int y3=blockIdx.y*blockDim.y+threadIdx.y+128-disp_max;//128-192
	//output[threadIdx.x][threadIdx.y]=0;
	if((y>disp_max+(kernel_size-1)/2+box_kernel)&&(y<cols-disp_max-(kernel_size-1)/2-box_kernel)&&(x>(kernel_size-1)/2+box_kernel+filterweight_kernel_h)&&(x<rows-(kernel_size-1)/2-box_kernel-filterweight_kernel_h))//y>disp_max，表示了左边disp_max个点不要了
	{
		//configure left shared memory
		sh_pic[threadIdx.x][threadIdx.y]=a[x-filterweight_kernel_h][y-filterweight_kernel_h];//因为共享内存上下左右各大了一行，而坐标原点在左上角，所以需要全部-1
		sh_disp[threadIdx.x][threadIdx.y]=b[x-filterweight_kernel_h][y-filterweight_kernel_h];
		if(threadIdx.x<(2*filterweight_kernel_h))//内存重复覆盖的问题？这里这个threadIdx.x是不是一定小于2,因为block_size_x是2，所以判断语句是不是不需要了？
		{
			sh_pic[threadIdx.x+block_size_x][threadIdx.y]=a[x+block_size_x-filterweight_kernel_h][y-filterweight_kernel_h];//相当于存了4行的数据，上下各拓宽了一行
			sh_disp[threadIdx.x+block_size_x][threadIdx.y]=b[x+block_size_x-filterweight_kernel_h][y-filterweight_kernel_h];
		}
		if(threadIdx.y<(2*filterweight_kernel_h))//这个是为了扩充右边界，因为上下左右各拓宽了一行
		{
			sh_pic[threadIdx.x][threadIdx.y+block_size_y]=a[x-filterweight_kernel_h][y+block_size_y-filterweight_kernel_h];
			sh_disp[threadIdx.x][threadIdx.y+block_size_y]=b[x-filterweight_kernel_h][y+block_size_y-filterweight_kernel_h];
		}
		if((threadIdx.x<2*filterweight_kernel_h)&&(threadIdx.y<2*filterweight_kernel_h))//这个是为了更新边框右下角的数据
		{
			sh_pic[threadIdx.x+block_size_x][threadIdx.y+block_size_y]=a[x+block_size_x-filterweight_kernel_h][y+block_size_y-filterweight_kernel_h];
			sh_disp[threadIdx.x+block_size_x][threadIdx.y+block_size_y]=b[x+block_size_x-filterweight_kernel_h][y+block_size_y-filterweight_kernel_h];
		}
		__syncthreads();

		float temp_pic_c[2*filterweight_kernel_h+1][2*filterweight_kernel_h+1];
		float temp_pic_a[2*filterweight_kernel_h+1][2*filterweight_kernel_h+1];//color+distance=all,temp_pic_d
		float temp_disp[2*filterweight_kernel_h+1][2*filterweight_kernel_h+1];

		int model1=255;
		int model2=255*2^8;
		int temp1_l=0;
		int temp1_r=0;
		int temp2_l=0;
		int temp2_r=0;
		int temp3_l=0;
		int temp3_r=0;


		temp1_r=(sh_pic[threadIdx.x+filterweight_kernel_h][threadIdx.y+filterweight_kernel_h])&model1;//这里1r存的是中心点的低8位值
		temp2_r=(sh_pic[threadIdx.x+filterweight_kernel_h][threadIdx.y+filterweight_kernel_h])&model2;//这里2r存的是中心点的中8位值
		temp2_r=temp2_r>>8;
		temp3_r=(sh_pic[threadIdx.x+filterweight_kernel_h][threadIdx.y+filterweight_kernel_h])>>16;//这里3r存的是中心点的高8位值
		for(int i=0;i<(2*filterweight_kernel_h+1);i++)
		{
			for(int j=0;j<(2*filterweight_kernel_h+1);j++)//关于颜色的地方需要改改！！RGB
			{
				temp1_l=(sh_pic[threadIdx.x+i][threadIdx.y+j])&model1;
				temp2_l=(sh_pic[threadIdx.x+i][threadIdx.y+j])&model2;
				temp2_l=temp2_l>>8;
				temp3_l=(sh_pic[threadIdx.x+i][threadIdx.y+j])>>16;
				temp_pic_c[i][j]=sqrt((float)((temp1_l-temp1_r)*(temp1_l-temp1_r)+(temp2_l-temp2_r)*(temp2_l-temp2_r)+(temp3_l-temp3_r)*(temp3_l-temp3_r)));
			}
		}
		for(int i=0;i<(2*filterweight_kernel_h+1);i++)
		{
			for(int j=0;j<(2*filterweight_kernel_h+1);j++)
			{
				temp_pic_a[i][j]=exp(-1*(temp_pic_c[i][j]/sigma_c)*(temp_pic_c[i][j]/sigma_c))*exp(-1*(temp_d[i][j]/sigma_p)*(temp_d[i][j]/sigma_p));//sigma_c和sigma_p需要定义
			}
		}
//		for(int i=0;i<(2*filterweight_kernel_h+1);i++)
//		{
//			for(int j=0;j<(2*filterweight_kernel_h+1);j++)
//			{
//				temp_disp[i][j]=temp_pic_a[i][j]*(float)sh_disp[threadIdx.x+i][threadIdx.y+j];//sigma_c和sigma_p需要定义
//			}
//		}
		//Middle filter
		float temp_disp_middle[(2*filterweight_kernel_h+1)*(2*filterweight_kernel_h+1)];
		float temp_pic_middle[(2*filterweight_kernel_h+1)*(2*filterweight_kernel_h+1)];
		for(int i=0;i<(2*filterweight_kernel_h+1);i++)
		{
			for(int j=0;j<(2*filterweight_kernel_h+1);j++)
			{
				temp_disp_middle[i*(2*filterweight_kernel_h+1)+j]=(float)sh_disp[threadIdx.x+i][threadIdx.y+j];
				temp_pic_middle[i*(2*filterweight_kernel_h+1)+j]=temp_pic_a[i][j];
			}
		}

		for (int j=0; j<(2*filterweight_kernel_h+1); ++j)
		    {
		        int min_middle=j;
		        for (int l=j+1; l<(2*filterweight_kernel_h+1)*(2*filterweight_kernel_h+1)-1; ++l)
		            if (temp_disp_middle[l] < temp_disp_middle[min_middle])
		            	min_middle=l;
		        //视差排序同时权值排序
		        const float bbtemp=temp_disp_middle[j];
		        temp_disp_middle[j]=temp_disp_middle[min_middle];
		        temp_disp_middle[min_middle]=bbtemp;

		        const float bbbtemp=temp_pic_middle[j];
		        temp_pic_middle[j]=temp_pic_middle[min_middle];
		        temp_pic_middle[min_middle]=bbbtemp;
		    }
		float SUM=0;
		for (int i=0;i<(2*filterweight_kernel_h+1)*(2*filterweight_kernel_h+1);i++)
		{
			SUM= SUM + temp_pic_middle[i];
		}
		SUM=SUM*MIDDLE_kernel_h;
		float Mid_SUM=0;
		for (int i=0;i<(2*filterweight_kernel_h+1);i++)
		{
			for (int j=0;j<(2*filterweight_kernel_h+1);j++)
			{
				Mid_SUM= Mid_SUM + temp_pic_middle[j+(2*filterweight_kernel_h+1)*i];
				if(Mid_SUM>=SUM)
				{
					output[x][y]= temp_disp_middle[j+(2*filterweight_kernel_h+1)*i];
					i=2*filterweight_kernel_h+1;
					break;
				}
			}
		}

//		for(int i=0;i<(2*filterweight_kernel_h+1); ++i)
//		{
//			for(int j=0;j<(2*filterweight_kernel_h+1);j++)
//			{
//				if(temp_disp[i][j]==temp_middle[filterweight_kernel_h])
//				{
//					output[x][y]= sh_disp[i+threadIdx.x][j+threadIdx.y];
//					i=2*filterweight_kernel_h+1;
//					break;
//				}
//			}
//		}
	}
}





int main()
{
	//hipSetDevice(0);
	//hipDeviceProp_t deviceProp;
	//hipGetDeviceProperties(&deviceProp, 0);
	//deviceProp.unifiedAddressing=0;

	dim3 threads(block_size_x,block_size_y);
	dim3 blocks(iDivUp(rows,block_size_x),iDivUp(cols,block_size_y));
	float temp_pic_d[2*filterweight_kernel+1][2*filterweight_kernel+1];
	float temp_pic_d_h[2*filterweight_kernel_h+1][2*filterweight_kernel_h+1];

	uint (*cpu_p1)[cols];
	uint (*cpu_p2)[cols];
	uint (*gpu_p1)[cols];
	uint (*gpu_p1_flip)[cols];
	uint (*gpu_p2)[cols];
	uint (*gpu_p2_flip)[cols];
	uchar (*gpu_p3)[cols];
	uchar (*gpu_p4)[cols];
	uchar (*gpu_p5)[cols];
	uchar (*gpu_p6)[cols];
	uchar (*dr)[cols];
	uchar (*dd)[cols];
	uchar (*dl)[cols];
	uchar (*du)[cols];
	uchar (*dlr)[cols];
	uchar (*dud)[cols];
	uchar (*gpu_p7)[cols];
	bool (*gpu_LR_error)[cols];
	bool (*gpu_LR_error_R)[cols];
	float (*temp_pic_disp)[2*filterweight_kernel+1];
	float (*temp_pic_disp_h)[2*filterweight_kernel_h+1];

	uchar (*gpu_p8)[cols];
	uchar (*gpu_p9)[cols];
	uchar (*gpu_p10)[cols];
	uchar (*gpu_p11)[cols];
	uchar (*gpu_p12)[cols];
	uchar (*gpu_p13)[cols];
	uchar (*gpu_p14)[cols];
	uchar (*gpu_p15)[cols];
	uchar (*gpu_p16)[cols];
	uchar (*gpu_p17)[cols];
	uchar (*gpu_p18)[cols];
	uchar (*gpu_p19)[cols];

	Mat im1,im2,im3,im4;
	im3.create(rows,cols,CV_8UC1);
	im4.create(rows,cols,CV_8UC1);
	im1=imread("im0.png");
	im2=imread("im1.png");

//	imshow("左图",im1);

	//锁页内存
	hipHostAlloc( (void**)&cpu_p1,rows*cols*sizeof(uint),hipHostMallocDefault);
	hipHostAlloc( (void**)&cpu_p2,rows*cols*sizeof(uint),hipHostMallocDefault);


	for(int x=0;x<rows;x++)
	{
		for(int y=0;y<cols;y++)
		{
			cpu_p1[x][y]=im1.at<Vec3b>(x,y)[0]+(im1.at<Vec3b>(x,y)[1]<<8)+(im1.at<Vec3b>(x,y)[2]<<16);//这个左移神马意思？
			cpu_p2[x][y]=im2.at<Vec3b>(x,y)[0]+(im2.at<Vec3b>(x,y)[1]<<8)+(im2.at<Vec3b>(x,y)[2]<<16);//读进来是char型，cpu_p1是int型
		}
	}


	for(int i=0;i<(2*filterweight_kernel+1);i++)
	{
		for(int j=0;j<(2*filterweight_kernel+1);j++)
		{
			temp_pic_d[i][j]=(i-filterweight_kernel)*(i-filterweight_kernel)+(j-filterweight_kernel)*(j-filterweight_kernel);
		}
	}
	for(int i=0;i<(2*filterweight_kernel_h+1);i++)
	{
		for(int j=0;j<(2*filterweight_kernel_h+1);j++)
		{
			temp_pic_d_h[i][j]=(i-filterweight_kernel_h)*(i-filterweight_kernel_h)+(j-filterweight_kernel_h)*(j-filterweight_kernel_h);
		}
	}



	//自动补位
	size_t pitch;
	hipMallocPitch((void **)&gpu_p1,&pitch,cols*sizeof(uint),rows);//这个pitch自动补位存放的是补位值，如果覆盖了以后就无法访问了
	hipMallocPitch((void **)&gpu_p2,&pitch,cols*sizeof(uint),rows);//我理解这么做是为了运行快
	hipMallocPitch((void **)&gpu_p1_flip,&pitch,cols*sizeof(uint),rows);//我理解这么做是为了运行快
	hipMallocPitch((void **)&gpu_p2_flip,&pitch,cols*sizeof(uint),rows);//我理解这么做是为了运行快
	hipMallocPitch((void **)&gpu_p3,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p4,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p5,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p6,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&temp,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&dr,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&dd,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&dl,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&du,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&dlr,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&dud,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p7,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_LR_error,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_LR_error_R,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p8,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p9,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p10,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p11,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p12,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p13,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&temp_pic_disp,&pitch,(2*filterweight_kernel+1)*sizeof(float),(2*filterweight_kernel+1));
	hipMallocPitch((void **)&temp_pic_disp_h,&pitch,(2*filterweight_kernel_h+1)*sizeof(float),(2*filterweight_kernel_h+1));
	hipMallocPitch((void **)&gpu_p14,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p15,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p16,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p17,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p18,&pitch,cols*sizeof(uchar),rows);
	hipMallocPitch((void **)&gpu_p19,&pitch,cols*sizeof(uchar),rows);

	hipMemcpyAsync(gpu_p1,cpu_p1,rows*cols*sizeof(uint),hipMemcpyHostToDevice);//数据传输阻塞和非阻塞，同步和异步？
	hipMemcpyAsync(gpu_p2,cpu_p2,rows*cols*sizeof(uint),hipMemcpyHostToDevice);
	hipMemcpyAsync(temp_pic_disp,temp_pic_d,(2*filterweight_kernel+1)*(2*filterweight_kernel+1)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpyAsync(temp_pic_disp_h,temp_pic_d_h,(2*filterweight_kernel_h+1)*(2*filterweight_kernel_h+1)*sizeof(float),hipMemcpyHostToDevice);
//	hipEvent_t start,stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord(start,0);
//
//	hipEventRecord(stop,0);
//	hipEventSynchronize(stop);
//	float time;
//	hipEventElapsedTime(&time,start,stop);
//	printf("Time is %fms\n",time);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	flip_kernel<<<blocks,threads>>>(gpu_p1,gpu_p1_flip);
	flip_kernel<<<blocks,threads>>>(gpu_p2,gpu_p2_flip);
	stereo_kernel<<<blocks,threads>>>(gpu_p1,gpu_p2,gpu_p3);
	stereo_kernel<<<blocks,threads>>>(gpu_p2_flip,gpu_p1_flip,gpu_p5);
	flip_kernel_char<<<blocks,threads>>>(gpu_p5,gpu_p6);

	Match_error<<<blocks,threads>>>(gpu_p3,gpu_p6,gpu_LR_error);
	near_match<<<blocks,threads>>>(	gpu_p3,gpu_p7,gpu_LR_error,dr,
									dl,du,dd,
									dlr,dud);
//
	Match_error<<<blocks,threads>>>(gpu_p6,gpu_p3,gpu_LR_error_R);
	near_match<<<blocks,threads>>>(	gpu_p6,gpu_p15,gpu_LR_error_R,dr,
									dl,du,dd,
									dlr,dud);


//	middle_filter<<<blocks,threads>>>(gpu_p7,gpu_p10,Middle_Radius);
	filter_weight<<<blocks,threads>>>(gpu_p1,gpu_p3,gpu_p13,temp_pic_disp);
	filter_weight<<<blocks,threads>>>(gpu_p2,gpu_p15,gpu_p14,temp_pic_disp);

	filter_weight_h<<<blocks,threads>>>(gpu_p1,gpu_p13,gpu_p17,temp_pic_disp_h);
	filter_weight_h<<<blocks,threads>>>(gpu_p2,gpu_p14,gpu_p16,temp_pic_disp_h);
	filter_weight_h<<<blocks,threads>>>(gpu_p2,gpu_p16,gpu_p18,temp_pic_disp_h);
	filter_weight_h<<<blocks,threads>>>(gpu_p2,gpu_p18,gpu_p19,temp_pic_disp_h);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	printf("Time is %fms\n",time);
//	box_filter(gpu_p7,gpu_p4,0,blocks,threads);
//	pic_show<<<blocks,threads>>>(gpu_p13,gpu_p11);
//	pic_show<<<blocks,threads>>>(gpu_p3,gpu_p12);





	hipMemcpy(im3.data,gpu_p17,rows*cols*sizeof(uchar),hipMemcpyDeviceToHost);
	hipMemcpy(im4.data,gpu_p19,rows*cols*sizeof(uchar),hipMemcpyDeviceToHost);
	//imshow("视差图l",im3);
	//imshow("视差图r",im4);
	imwrite("disp_l.bmp",im3);
	imwrite("disp_r.bmp",im4);
	waitKey(0);
	return 0;
}
